
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N_DIM 2048
#define M_DIM 8
#define SHARED 1

__constant__ float Mask[M_DIM];

__global__ void conv_1(float *M, float *P, float *N, int Mask_Width, int vector_Width){
	int tid= blockIdx.x * blockDim.x +threadIdx.x;

	float Pvalue = 0;

	int N_start_point = tid - (Mask_Width/2);

	for(int t=0; t<Mask_Width; ++t){
		if(N_start_point +t >=0 && N_start_point +t < vector_Width){
			Pvalue+= Mask[t] * N[N_start_point+t];
		}
	}
	P[tid] = Pvalue;
}

__global__ void conv_1_s(float *M, float *P, float *N, int Mask_Width, int vector_Width){
	int tid= blockIdx.x * blockDim.x +threadIdx.x;
	
	//extern __shared__ float s[];
	__shared__ int s[512]; 
	float Pvalue = 0;
	
	s[0] =0;
	s[threadIdx.x] = N[tid];
	
	int base = 0; // tid - threadIdx.x;
	int roof = blockDim.x; //tid - threadIdx.x + blockDim.x;
	
	__syncthreads();

	int N_start_point = tid - (Mask_Width/2);
	int N_start_point_shared = threadIdx.x - (Mask_Width/2);
	//printf("%d - %d ----- %d - %d\n", base, roof, N_start_point, N_start_point_shared);

	for(int t=0; t<Mask_Width; ++t){
		/*if(tid == 30){
			printf("%d - %d - %d ----- %d - %d ---- %d - %d\n",threadIdx.x, base, roof, N_start_point, N_start_point_shared, N_start_point_shared +t, N_start_point +t);
			printf("on mask: %d + %d >= 0 && %d + %d < %d\n", N_start_point,t,N_start_point,t, vector_Width);
			printf("on shared: %d + %d >= %d && %d + %d < %d\n", N_start_point_shared,t, base, N_start_point_shared,t, roof);
		}*/
		if(N_start_point +t >=0 && N_start_point +t < vector_Width){
			//if(tid == 30) printf("hey\n");
			if(N_start_point_shared +t >=base && N_start_point_shared + t<roof){
				//if(tid==30) printf("convolving %.2f from shared mem\n",s[N_start_point_shared+t]);
				Pvalue+= Mask[t] * s[N_start_point_shared+t];
			}else{
				//if(tid==30) printf("convolving %.2f from global mem\n",s[N_start_point_shared+t]);
				Pvalue+= Mask[t] * N[N_start_point+t];
			}		
		}
	}
	//if(tid==30) printf("final val is %.2f\n",Pvalue);
	P[tid] = Pvalue;
}


int main(void){
	float * N, *M, *P;
	float *d_N, *d_M, *d_P;

	N = (float *) malloc(N_DIM*sizeof(float));
	M = (float *) malloc(M_DIM*sizeof(float));
	P = (float *) malloc(N_DIM*sizeof(float));


	
	for(int t =0; t<N_DIM; ++t){
		N[t] =1;
	}
	for(int t =0; t<M_DIM; ++t){
		M[t] =1;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(Mask), M, sizeof(float)*M_DIM);

	hipMalloc((void**)&d_N, N_DIM*sizeof(float));
	hipMalloc((void**)&d_M, M_DIM*sizeof(float));
	hipMalloc((void**)&d_P, N_DIM*sizeof(float));

	hipMemcpy(d_N, N, N_DIM*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_M, M, M_DIM*sizeof(float), hipMemcpyHostToDevice);
	
	int threads = 512;
	int blocks = N_DIM/threads +1;
	//int threads = 16;
	//int blocks = 8;
#if SHARED
	printf("shared\n");
	conv_1_s<<<blocks,threads>>>(d_M, d_P, d_N, M_DIM, N_DIM);
#else
	printf("not shared\n");
	conv_1<<<blocks,threads>>>(d_M, d_P, d_N, M_DIM, N_DIM);
#endif
	hipMemcpy(P, d_P, N_DIM*sizeof(float), hipMemcpyDeviceToHost);
	/*for(int t=0; t<N_DIM; ++t){
		printf("%d: %.2f\n",t, P[t]);

	}*/

	hipFree(d_N);
	hipFree(d_M);
	hipFree(d_P);

	free(N);
	free(M);
	free(P);

}
