
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 2048

__global__ void vectorAdd(int *a, int *b, int *c){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	c[tid]=a[tid]+b[tid];
	//printf("tid: %d done, res: %d\n",tid, c[tid]);

}

int main(void){
	printf("starting vectorAdd\n");
	int *a, *b, *c;
	int *dev_a, *dev_b, *dev_c;

	//allocate and init cpu mem
	a =(int*) malloc( N* sizeof(int));
	b =(int*) malloc( N* sizeof(int));
	c =(int*) malloc( N* sizeof(int));

	for(int t=0; t<N; ++t){
		a[t]=t;
		b[t]=10;
		c[t]=0;
	}

	//allocate GPU mem
	hipMalloc((void**)&dev_a, N*sizeof(int));
	hipMalloc((void**)&dev_b, N*sizeof(int));
	hipMalloc((void**)&dev_c, N*sizeof(int));

	//copy the cpu arrays into gpu ones
	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	//exec
	int n_blocks = N/1024;
	if(n_blocks ==0){
		n_blocks = 1;
	}
	vectorAdd<<<n_blocks,N/n_blocks>>>(dev_a, dev_b, dev_c);

	//copy in c
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

	//check
	printf("check\n");
	int bad=0;
	for(int t=0; t<N && bad==0; ++t){
		if(c[t]!=a[t]+b[t]){
			printf("bad\n");
			bad=1;
		}
	}
        printf("\nend\n");
	//frees
	free(a);
	free(b);
	free(c);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	

	hipDeviceReset();
	return 0;
}
